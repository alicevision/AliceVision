#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2018 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#include <hip/hip_runtime.h>

#include "gauss_filter.hpp"
#include <aliceVision/depthMap/cuda/deviceCommon/device_operators.cuh>
#include <aliceVision/depthMap/cuda/planeSweeping/host_utils.h>
#include <aliceVision/depthMap/cuda/deviceCommon/device_utils.cuh>
#include <aliceVision/depthMap/cuda/commonStructures.hpp>


namespace aliceVision {
namespace depthMap {

/*********************************************************************************
* global / constant data structures
*********************************************************************************/
std::set<int>                 d_gaussianArrayInitialized;
__device__ __constant__ int   d_gaussianArrayOffset[MAX_CONSTANT_GAUSS_SCALES];
__device__ __constant__ float d_gaussianArray[MAX_CONSTANT_GAUSS_MEM_SIZE];

/*********************************************************************************
 * kernel forward declarations
 *********************************************************************************/
__global__ void downscale_gauss_smooth_lab_kernel(
    hipTextureObject_t rc_tex,
    CudaRGBA* texLab, int texLab_p,
    int width, int height, int scale, int radius);

/*********************************************************************************
 * exported host function
 *********************************************************************************/
__host__ void ps_create_gaussian_arr( int deviceId, int scales ) // float delta, int radius)
{
    if( scales >= MAX_CONSTANT_GAUSS_SCALES )
    {
        throw std::runtime_error( "Programming error: too few scales pre-computed for Gaussian kernels. Enlarge and recompile." );
    }

    hipError_t err;

    if( d_gaussianArrayInitialized.find( deviceId ) != d_gaussianArrayInitialized.end() ) return;

    d_gaussianArrayInitialized.insert( deviceId );

    int*   h_gaussianArrayOffset;
    float* h_gaussianArray;
    err = hipHostMalloc( &h_gaussianArrayOffset, MAX_CONSTANT_GAUSS_SCALES * sizeof(int) );
    THROW_ON_CUDA_ERROR(err, "Failed to allocate " << MAX_CONSTANT_GAUSS_SCALES * sizeof(int) << " of CUDA host memory."); 

    err = hipHostMalloc( &h_gaussianArray,       MAX_CONSTANT_GAUSS_MEM_SIZE * sizeof(float) );
    THROW_ON_CUDA_ERROR(err, "Failed to allocate " << MAX_CONSTANT_GAUSS_MEM_SIZE * sizeof(float) << " of CUDA host memory.");

    int sum_sizes = 0;
    for( int scale=0; scale<MAX_CONSTANT_GAUSS_SCALES; scale++ )
    {
        h_gaussianArrayOffset[scale] = sum_sizes;
        const int   radius = scale + 1;
        const int   size   = 2 * radius + 1;
        sum_sizes += size;
    }

    if( sum_sizes >= MAX_CONSTANT_GAUSS_MEM_SIZE )
    {
        throw std::runtime_error( "Programming error: too little memory allocated for " 
            + std::to_string(MAX_CONSTANT_GAUSS_SCALES) + " Gaussian kernels. Enlarge and recompile." );
    }

    for( int scale=0; scale<MAX_CONSTANT_GAUSS_SCALES; scale++ )
    {
        const int   radius = scale + 1;
        const float delta  = 1.0f;
        const int   size   = 2 * radius + 1;

        for( int idx=0; idx<size; idx++ )
        {
            int x = idx - radius;
            h_gaussianArray[h_gaussianArrayOffset[scale]+idx] = expf(-(x * x) / (2 * delta * delta));
        }

        // generate gaussian array
    }


    // create cuda array
    err = hipMemcpyToSymbol(HIP_SYMBOL( d_gaussianArrayOffset),
                              h_gaussianArrayOffset,
                              MAX_CONSTANT_GAUSS_SCALES * sizeof(int), 0, hipMemcpyHostToDevice);

    THROW_ON_CUDA_ERROR(err, "Failed to move Gaussian filter to symbol.");

    err = hipMemcpyToSymbol(HIP_SYMBOL( d_gaussianArray),
                              h_gaussianArray,
                              sum_sizes * sizeof(float), 0, hipMemcpyHostToDevice);

    THROW_ON_CUDA_ERROR(err, "Failed to move Gaussian filter to symbol." );

    hipHostFree( h_gaussianArrayOffset );
    hipHostFree( h_gaussianArray );
}

__host__ void ps_downscale_gauss( Pyramid& pyramid,
                                  int scale,
                                  int w, int h, int radius,
                                  hipStream_t stream )
{
    const dim3 block(32, 2, 1);
    const dim3 grid(divUp(w / (scale + 1), block.x), divUp(h / (scale + 1), block.y), 1);

    downscale_gauss_smooth_lab_kernel
        <<<grid, block, 0, stream>>>
        ( pyramid[0].tex,
          pyramid[scale].arr->getBuffer(),
          pyramid[scale].arr->getPitch(),
          w / (scale + 1), h / (scale + 1), scale + 1,
          radius
          );
}
/*********************************************************************************
 * kernel definitions
 *********************************************************************************/

/* This is a bad Gaussian filter implementation - the Gaussian filter is separable. */
__global__ void downscale_gauss_smooth_lab_kernel(
    hipTextureObject_t rc_tex,
    CudaRGBA* texLab, int texLab_p,
    int width, int height, int scale, int radius)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if((x < width) && (y < height))
    {
        float s = (float)scale * 0.5f;
        float4 t = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        float sum = 0.0f;
        for(int i = -radius; i <= radius; i++)
        {
            for(int j = -radius; j <= radius; j++)
            {
                float4 curPix = tex2D_float4(rc_tex, (float)(x * scale + j) + s,
                                               (float)(y * scale + i) + s);
                float factor = getGauss( scale-1, i + radius )
                             * getGauss( scale-1, j + radius ); // domain factor
                t = t + curPix * factor;
                sum += factor;
            }
        }
        t.x = t.x / sum;
        t.y = t.y / sum;
        t.z = t.z / sum;
        t.w = t.w / sum;

        CudaRGBA& out = BufPtr<CudaRGBA>(texLab, texLab_p).at(x,y);
        out.x = t.x;
        out.y = t.y;
        out.z = t.z;
        out.w = t.w;
    }
}

__device__ void cuda_swap_float(float& a, float& b)
{
    float temp = a;
    a = b;
    b = temp;
}

/**
* @warning: use an hardcoded buffer size, so max radius value is 3.
*/
__global__ void medianFilter3_kernel(
    hipTextureObject_t tex,
    float* texLab, int texLab_p,
    int width, int height,
    int scale)
{
    const int radius = 3;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x >= width - radius) || (y >= height - radius) ||
        (x < radius) || (y < radius))
        return;

    const int filterWidth = radius * 2 + 1;
    const int filterNbPixels = filterWidth * filterWidth;

    float buf[filterNbPixels]; // filterNbPixels

    // Assign masked values to buf
    for (int yi = 0; yi < filterWidth; ++yi)
    {
        for (int xi = 0; xi < filterWidth; ++xi)
        {
            float pix = tex2D<float>(tex, x + xi - radius, y + yi - radius);
            buf[yi * filterWidth + xi] = pix;
        }
    }

    // Calculate until we get the median value
    for (int k = 0; k < filterNbPixels; ++k) // (filterNbPixels + 1) / 2
        for (int l = 0; l < filterNbPixels; ++l)
            if (buf[k] < buf[l])
                cuda_swap_float(buf[k], buf[l]);

    BufPtr<float>(texLab, texLab_p).at(x, y) = buf[radius * filterWidth + radius];
}


__host__ void ps_medianFilter3(
    hipTextureObject_t tex,
    CudaDeviceMemoryPitched<float, 2>& img)
{
    int scale = 1;
    const dim3 block(32, 2, 1);
    const dim3 grid(divUp(img.getSize()[0], block.x), divUp(img.getSize()[1], block.y), 1);

    medianFilter3_kernel
        <<<grid, block>>>
        (tex,
            img.getBuffer(), img.getPitch(),
            img.getSize()[0], img.getSize()[1],
            scale
            );
}


} // namespace depthMap
} // namespace aliceVision

