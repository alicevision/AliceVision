#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2017 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

namespace aliceVision {
namespace depthMap {

/**
 * @param[in] s: iteration over nSamplesHalf
 */
__global__ void fuse_computeGaussianKernelVotingSampleMap_kernel(float* out_gsvSampleMap, int out_gsvSampleMap_p,
                                                                 float2* depthSimMap, int depthSimMap_p,
                                                                 float2* midDepthPixSizeMap, int midDepthPixSizeMap_p,
                                                                 int width, int height, float s, int idCam,
                                                                 float samplesPerPixSize, float twoTimesSigmaPowerTwo)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if((x >= 0) && (y >= 0) && (x < width) && (y < height))
    {
        float2 midDepthPixSize = *get2DBufferAt(midDepthPixSizeMap, midDepthPixSizeMap_p, x, y);
        float2 depthSim = *get2DBufferAt(depthSimMap, depthSimMap_p, x, y);
        float* out_gsvSample_ptr = get2DBufferAt(out_gsvSampleMap, out_gsvSampleMap_p, x, y);
        float gsvSample = (idCam == 0) ? 0.0f : *out_gsvSample_ptr;

        if((midDepthPixSize.x > 0.0f) && (depthSim.x > 0.0f))
        {
            float depthStep = midDepthPixSize.y / samplesPerPixSize;
            float i = (midDepthPixSize.x - depthSim.x) / depthStep;
            float sim = -sigmoid(0.0f, 1.0f, 0.7f, -0.7f, depthSim.y);
            gsvSample += sim * expf(-((i - s) * (i - s)) / twoTimesSigmaPowerTwo);
        };
        *out_gsvSample_ptr = gsvSample;
    };
}


__global__ void fuse_updateBestGaussianKernelVotingSampleMap_kernel(float2* bestGsvSampleMap, int bestGsvSampleMap_p,
                                                                    float* gsvSampleMap, int gsvSampleMap_p, int width,
                                                                    int height, float s, int id)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if((x >= 0) && (y >= 0) && (x < width) && (y < height))
    {
        float gsvSampleX = *get2DBufferAt(gsvSampleMap, gsvSampleMap_p, x, y);
        float2* bestGsvSample_ptr = get2DBufferAt(bestGsvSampleMap, bestGsvSampleMap_p, x, y);
        if(id == 0 || gsvSampleX < bestGsvSample_ptr->x)
            *bestGsvSample_ptr = make_float2(gsvSampleX, s);
    };
}

__global__ void fuse_computeFusedDepthSimMapFromBestGaussianKernelVotingSampleMap_kernel(
    float2* oDepthSimMap, int oDepthSimMap_p, float2* bestGsvSampleMap, int bestGsvSampleMap_p,
    float2* midDepthPixSizeMap, int midDepthPixSizeMap_p, int width, int height, float samplesPerPixSize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if((x >= 0) && (y >= 0) && (x < width) && (y < height))
    {
        float2 bestGsvSample = *get2DBufferAt(bestGsvSampleMap, bestGsvSampleMap_p, x, y);
        float2 midDepthPixSize = *get2DBufferAt(midDepthPixSizeMap, midDepthPixSizeMap_p, x, y);
        float depthStep = midDepthPixSize.y / samplesPerPixSize;

        // normalize similarity to -1,0
        // figure; t = -5.0:0.01:0.0; plot(t,sigmoid(0.0,-1.0,6.0,-0.4,t,0));
        //bestGsvSample.x = sigmoid(0.0f, -1.0f, 6.0f, -0.4f, bestGsvSample.x);
        float2* oDepthSim = get2DBufferAt(oDepthSimMap, oDepthSimMap_p, x, y);
        if(midDepthPixSize.x <= 0.0f)
            *oDepthSim = make_float2(-1.0f, 1.0f);
        else
            *oDepthSim = make_float2(midDepthPixSize.x - bestGsvSample.y * depthStep, bestGsvSample.x);
    };
}

__global__ void fuse_getOptDeptMapFromOPtDepthSimMap_kernel(float* optDepthMap, int optDepthMap_p,
                                                            float2* optDepthMapSimMap, int optDepthMapSimMap_p,
                                                            int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if((x >= 0) && (y >= 0) && (x < width) && (y < height))
    {
        *get2DBufferAt(optDepthMap, optDepthMap_p, x, y) = get2DBufferAt(optDepthMapSimMap, optDepthMapSimMap_p, x, y)->x;
    };
}
/**
 * @return (smoothStep, energy)
 */
__device__ float2 getCellSmoothStepEnergy(const int2& cell0)
{
    float2 out = make_float2(0.0f, 180.0f);

    // Get pixel depth from the depth texture
    float d0 = tex2D(depthsTex, cell0.x, cell0.y);

    // Early exit: depth is <= 0
    if(d0 <= 0.0f)
        return out;
    
    // Consider the neighbor pixels
    int2 cellL = cell0 + make_int2(0, -1);	// Left
    int2 cellR = cell0 + make_int2(0, 1);	// Right
    int2 cellU = cell0 + make_int2(-1, 0);	// Up
    int2 cellB = cell0 + make_int2(1, 0);	// Bottom

    // Get associated depths from depth texture
    float dL = tex2D(depthsTex, cellL.x, cellL.y);
    float dR = tex2D(depthsTex, cellR.x, cellR.y);
    float dU = tex2D(depthsTex, cellU.x, cellU.y);
    float dB = tex2D(depthsTex, cellB.x, cellB.y);

    // Get associated 3D points
    float3 p0 = get3DPointForPixelAndDepthFromRC(cell0, d0);
    float3 pL = get3DPointForPixelAndDepthFromRC(cellL, dL);
    float3 pR = get3DPointForPixelAndDepthFromRC(cellR, dR);
    float3 pU = get3DPointForPixelAndDepthFromRC(cellU, dU);
    float3 pB = get3DPointForPixelAndDepthFromRC(cellB, dB);

    // Compute the average point based on neighbors (cg)
    float3 cg = make_float3(0.0f, 0.0f, 0.0f);
    float n = 0.0f;

    if(dL > 0.0f) { cg = cg + pL; n++; }
    if(dR > 0.0f) { cg = cg + pR; n++; }
    if(dU > 0.0f) { cg = cg + pU; n++; }
    if(dB > 0.0f) { cg = cg + pB; n++; }
    
    // If we have at least one valid depth
    if(n > 1.0f)
    {
        cg = cg / n; // average of x, y, depth
        float3 vcn = sg_s_r.C - p0;
        normalize(vcn);
        // pS: projection of cg on the line from p0 to camera
        float3 pS = closestPointToLine3D(cg, p0, vcn);
        // keep the depth difference between pS and p0 as the smoothing step
        out.x = size(sg_s_r.C - pS) - d0;
    }

    float e = 0.0f;
    n = 0.0f;

    if(dL > 0.0f && dR > 0.0f)
    {
        // Large angle between neighbors == flat area => low energy
        // Small angle between neighbors == non-flat area => high energy
        e = fmaxf(e, (180.0f - angleBetwABandAC(p0, pL, pR)));
        n++;
    }
    if(dU > 0.0f && dB > 0.0f)
    {
        e = fmaxf(e, (180.0f - angleBetwABandAC(p0, pU, pB)));
        n++;
    }
    // The higher the energy, the less flat the area
    if(n > 0.0f)
        out.y = e;

    return out;
}

__global__ void fuse_optimizeDepthSimMap_kernel(float2* out_optDepthSimMap, int optDepthSimMap_p,
                                                float2* midDepthPixSizeMap, int midDepthPixSizeMap_p,
                                                float2* fusedDepthSimMap, int fusedDepthSimMap_p, int width, int height,
                                                int iter, float samplesPerPixSize, int yFrom)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int2 pix = make_int2(x, y);

    if((x >= 0) && (y >= 0) && (x < width) && (y < height))
    {
        float2 midDepthPixSize = *get2DBufferAt(midDepthPixSizeMap, midDepthPixSizeMap_p, x, y);
        float2 fusedDepthSim = *get2DBufferAt(fusedDepthSimMap, fusedDepthSimMap_p, x, y);
        float2* out_optDepthSim_ptr = get2DBufferAt(out_optDepthSimMap, optDepthSimMap_p, x, y);
        float2 out_optDepthSim = (iter == 0) ? make_float2(midDepthPixSize.x, fusedDepthSim.y) : *out_optDepthSim_ptr;

        float depthOpt = out_optDepthSim.x;

        if(depthOpt > 0.0f)
        {
            float2 depthSmoothStepEnergy = getCellSmoothStepEnergy(pix);
            float depthSmoothStep = depthSmoothStepEnergy.x;
            if(depthSmoothStep < 0.0f)
            {
                depthSmoothStep = -fminf(fabsf(depthSmoothStep), midDepthPixSize.y / 10.0f);
            }
            else
            {
                depthSmoothStep = +fminf(fabsf(depthSmoothStep), midDepthPixSize.y / 10.0f);
            };

            float depthPhotoStep = fusedDepthSim.x - depthOpt;
            if(depthPhotoStep < 0.0f)
            {
                depthPhotoStep = -fminf(fabsf(depthPhotoStep), midDepthPixSize.y / 10.0f);
            }
            else
            {
                depthPhotoStep = +fminf(fabsf(depthPhotoStep), midDepthPixSize.y / 10.0f);
            };

            float depthVisStep = midDepthPixSize.x - depthOpt;

            float depthSmoothVal = depthSmoothStepEnergy.y;
            float depthPhotoStepVal = fusedDepthSim.y;

            float varianceGray = 255.0f*tex2D(r4tex, (float)x + 0.5f, (float)(y + yFrom) + 0.5f).w;

            // archive: 
            // float varianceGrayAndleWeight = sigmoid2(5.0f, 60.0f, 10.0f, 5.0f, varianceGray);
            // 0.6:
            float varianceGrayAndleWeight = sigmoid2(5.0f, 30.0f, 40.0f, 20.0f, varianceGray);

            // archive: 
            // float simWeight = -depthPhotoStepVal; // must be from 0 to 1=from worst=0 to best=1 ... it is from -1 to 0
            // 0.6:
            float simWeight = sigmoid(0.0f, 1.0f, 0.7f, -0.7f, depthPhotoStepVal);

            // archive: 
            // float photoWeight = sigmoid(0.0f, 1.0f, 60.0f, varianceGrayAndleWeight, depthSmoothVal);
            // 0.6:
            float photoWeight = sigmoid(0.0f, 1.0f, 30.0f, varianceGrayAndleWeight, depthSmoothVal);

            float smoothWeight = 1.0f - photoWeight;
            float visWeight = 1.0f - sigmoid(0.0f, 1.0f, 10.0f, 17.0f, fabsf(depthVisStep / midDepthPixSize.y));

            float depthOptStep = visWeight*depthVisStep + (1.0f - visWeight)*(photoWeight*simWeight*depthPhotoStep + smoothWeight*depthSmoothStep);

            out_optDepthSim.x = depthOpt + depthOptStep;

            // archive: 
            // optDepthSim.y = -photoWeight * simWeight
            // 0.6:
            out_optDepthSim.y = (1.0f - visWeight)*photoWeight*simWeight*depthPhotoStepVal + (1.0f - visWeight)*smoothWeight*(depthSmoothVal / 20.0f);
        };

        *out_optDepthSim_ptr = out_optDepthSim;
    };
}

} // namespace depthMap
} // namespace aliceVision
