#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2017 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#include <aliceVision/depthMap/cuda/commonStructures.hpp>
#include <aliceVision/depthMap/cuda/deviceCommon/device_color.cu>
#include <aliceVision/depthMap/cuda/deviceCommon/device_patch_es.cu>
// #include <aliceVision/depthMap/cuda/deviceCommon/device_eig33.cu>
#include <aliceVision/depthMap/cuda/planeSweeping/device_code.cu>
#include <aliceVision/depthMap/cuda/planeSweeping/device_code_refine.cu>
#include <aliceVision/depthMap/cuda/planeSweeping/device_code_volume.cu>
#include <aliceVision/depthMap/cuda/planeSweeping/device_code_fuse.cu>
#include <aliceVision/depthMap/cuda/planeSweeping/plane_sweeping_cuda.hpp>
#include <aliceVision/depthMap/cuda/planeSweeping/host_utils.h>
#include <aliceVision/depthMap/cuda/images/gauss_filter.hpp>

#include <hip/hip_math_constants.h>

#include <iostream>
#include <algorithm>
#include <map>
#include <array>

namespace aliceVision {
namespace depthMap {

// Macro for checking cuda errors
#define CHECK_CUDA_ERROR()                                                    \
    if(hipError_t err = hipGetLastError())                                  \
    {                                                                         \
        fprintf(stderr, "\n\nCUDAError: %s\n", hipGetErrorString(err));      \
        fprintf(stderr, "  file:       %s\n", __FILE__);                      \
        fprintf(stderr, "  function:   %s\n", __FUNCTION__);                  \
        fprintf(stderr, "  line:       %d\n\n", __LINE__);                    \
        std::stringstream s;                                                  \
        s << "\n  CUDA Error: " << hipGetErrorString(err)                    \
          << "\n  file:       " << __FILE__                                   \
          << "\n  function:   " << __FUNCTION__                               \
          << "\n  line:       " << __LINE__ << "\n";                          \
        throw std::runtime_error(s.str());                                    \
    }

#define ALICEVISION_CU_PRINT_DEBUG(a) \
    std::cerr << a << std::endl;

#define ALICEVISION_CU_PRINT_ERROR(a) \
    std::cerr << a << std::endl;

__host__ float3 ps_M3x3mulV3(const float* M3x3, const float3& V)
{
    return make_float3(M3x3[0] * V.x + M3x3[3] * V.y + M3x3[6] * V.z, M3x3[1] * V.x + M3x3[4] * V.y + M3x3[7] * V.z,
                       M3x3[2] * V.x + M3x3[5] * V.y + M3x3[8] * V.z);
}

__host__ void ps_normalize(float3& a)
{
    float d = sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
    a.x /= d;
    a.y /= d;
    a.z /= d;
}

void pr_printfDeviceMemoryInfo()
{
    size_t iavail;
    size_t itotal;
    hipMemGetInfo(&iavail, &itotal);
    size_t iused = itotal - iavail;

    double avail = double(iavail) / (1024.0 * 1024.0);
    double total = double(itotal) / (1024.0 * 1024.0);
    double used = double(iused) / (1024.0 * 1024.0);

    int CUDAdeviceNo;
    hipGetDevice(&CUDAdeviceNo);

    printf("Device %i memory - used: %f, free: %f, total: %f\n", CUDAdeviceNo, used, avail, total);
}

__host__ void ps_initCameraMatrix( CameraStructBase& base )
{
    float3 z;
    z.x = 0.0f;
    z.y = 0.0f;
    z.z = 1.0f;
    base.ZVect = ps_M3x3mulV3(base.iR, z);
    ps_normalize(base.ZVect);

    float3 y;
    y.x = 0.0f;
    y.y = 1.0f;
    y.z = 0.0f;
    base.YVect = ps_M3x3mulV3(base.iR, y);
    ps_normalize(base.YVect);

    float3 x;
    x.x = 1.0f;
    x.y = 0.0f;
    x.z = 0.0f;
    base.XVect = ps_M3x3mulV3(base.iR, x);
    ps_normalize(base.XVect);
}

int ps_listCUDADevices(bool verbose)
{
    int num_gpus = 0; // number of CUDA GPUs

    // determine the number of CUDA capable GPUs
    hipError_t err = hipGetDeviceCount(&num_gpus);
    CHECK_CUDA_ERROR();
    if(err != hipSuccess)
    {
        printf("Error getting cuda device count");
        return 0;
    }

    if(num_gpus < 1)
    {
        printf("ERROR: no CUDA capable devices detected");
        return 0;
    }

    if(verbose == true)
    {
        // display CPU and GPU configuration
        printf("number of CUDA devices:\t%d\n", num_gpus);
        for(int i = 0; i < num_gpus; i++)
        {
            hipDeviceProp_t dprop;
            hipGetDeviceProperties(&dprop, i);
            printf("   %d: %s\n", i, dprop.name);
        }
    }

    return num_gpus;
}

int ps_deviceAllocate(Pyramid& pyramid, int width, int height, int scales )
{
    int bytesAllocated = 0;

    pyramid.resize(scales);

    for(int s = 0; s < scales; s++)
    {
        int w = width / (s + 1);
        int h = height / (s + 1);
        // printf("ps_deviceAllocate: CudaDeviceMemoryPitched: [c%i][s%i] %ix%i\n", c, s, w, h);
        pyramid[s].arr = new CudaDeviceMemoryPitched<CudaRGBA, 2>(CudaSize<2>(w, h));
        bytesAllocated += pyramid[s].arr->getBytesPadded();

        hipTextureDesc  tex_desc;
        memset(&tex_desc, 0, sizeof(hipTextureDesc));
        tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
        tex_desc.addressMode[0]   = hipAddressModeClamp;
        tex_desc.addressMode[1]   = hipAddressModeClamp;
        tex_desc.addressMode[2]   = hipAddressModeClamp;
#if defined(ALICEVISION_DEPTHMAP_TEXTURE_USE_UCHAR) && defined(ALICEVISION_DEPTHMAP_TEXTURE_USE_INTERPOLATION)
        tex_desc.readMode = hipReadModeNormalizedFloat; // uchar to float [0:1], see tex2d_float4 function
#else
        tex_desc.readMode = hipReadModeElementType;
#endif
#ifdef ALICEVISION_DEPTHMAP_TEXTURE_USE_INTERPOLATION
        // with subpixel interpolation (can have a large performance impact on some graphic cards)
        // but could be critical for quality during SGM in small resolution
        tex_desc.filterMode = hipFilterModeLinear;
#else
        // without interpolation
        tex_desc.filterMode = hipFilterModePoint;
#endif

        hipResourceDesc res_desc;
        res_desc.resType = hipResourceTypePitch2D;
        res_desc.res.pitch2D.desc = hipCreateChannelDesc<CudaRGBA>();
        res_desc.res.pitch2D.devPtr       = pyramid[s].arr->getBuffer();
        res_desc.res.pitch2D.width        = pyramid[s].arr->getSize()[0];
        res_desc.res.pitch2D.height       = pyramid[s].arr->getSize()[1];
        res_desc.res.pitch2D.pitchInBytes = pyramid[s].arr->getPitch();

        hipError_t err = hipCreateTextureObject( &pyramid[s].tex, &res_desc, &tex_desc, 0 );
        THROW_ON_CUDA_ERROR( err, "Failed to bind texture object to cam array" );
    }

    return bytesAllocated;
}

void ps_deviceDeallocate( Pyramid& pyramid, int scales )
{
    for( TexturedArray& entry : pyramid )
    {
        delete entry.arr;
        hipDestroyTextureObject( entry.tex );
    }
    pyramid.clear();
}

void ps_testCUDAdeviceNo(int CUDAdeviceNo)
{
    int myCUDAdeviceNo;
    hipGetDevice(&myCUDAdeviceNo);
    if(myCUDAdeviceNo != CUDAdeviceNo)
    {
        printf("WARNING different device %i %i\n", myCUDAdeviceNo, CUDAdeviceNo);
    }
}

// void ps_device_updateCam( const CameraStruct& cam, int CUDAdeviceNo,
//                           int scales, int w, int h)
void ps_device_fillPyramidFromHostFrame( Pyramid& pyramid,
                          CudaHostMemoryHeap<CudaRGBA, 2>* host_frame,
                          int scales, int w, int h,
                          hipStream_t stream )
{
    ALICEVISION_CU_PRINT_DEBUG(std::endl
              << "Calling " << __FUNCTION__ << std::endl
              << "    for " << scales << " scales"
              << ", w: " << w << ", h: " << h
              << std::endl);

    {
        /* copy texture's data from host to device */
        pyramid[0].arr->copyFrom( *host_frame, stream );

        const dim3 block(32, 2, 1);
        const dim3 grid(divUp(w, block.x), divUp(h, block.y), 1);
        ALICEVISION_CU_PRINT_DEBUG("rgb2lab_kernel: block=(" << block.x << ", " << block.y << ", " << block.z << "), grid=(" << grid.x << ", " << grid.y << ", " << grid.z << ")");

        /* in-place color conversion into CIELAB */
        rgb2lab_kernel<<<grid, block, 0, stream>>>(
            pyramid[0].arr->getBuffer(), pyramid[0].arr->getPitch(),
            w, h);
        CHECK_CUDA_ERROR();
    }

    /* For each scale, create a Gaussian-filtered and scaled version of the
     * initial texture */
    for(int scale = 1; scale < scales; ++scale)
    {
        const int radius = scale + 1;
        // const int sWidth = w / (scale + 1);
        // const int sHeight = h / (scale + 1);
        // ALICEVISION_CU_PRINT_DEBUG("Create downscaled image for camera id " << camId << " at scale " << scale << ": " << sWidth << "x" << sHeight);

        // const dim3 block(32, 2, 1);
        // const dim3 grid(divUp(sWidth, block.x), divUp(sHeight, block.y), 1);
        // ALICEVISION_CU_PRINT_DEBUG("ps_downscale_gauss: block=(" << block.x << ", " << block.y << ", " << block.z << "), grid=(" << grid.x << ", " << grid.y << ", " << grid.z << ")");

        ps_downscale_gauss(pyramid, scale, w, h, radius, stream);
        CHECK_CUDA_ERROR();
    }

    CHECK_CUDA_ERROR();
}


/**
 * @param[inout] d_volSimT similarity volume
 */
void ps_aggregatePathVolume(
    CudaDeviceMemoryPitched<TSim, 3>& d_volAgr,
    const CudaDeviceMemoryPitched<TSim, 3>& d_volSim,
    const CudaSize<3>& volDim,
    const CudaSize<3>& axisT,
    hipTextureObject_t rc_tex, 
    const SgmParams& sgmParams,
    bool invY, int filteringIndex)
{
    const size_t volDimX = volDim[axisT[0]];
    const size_t volDimY = volDim[axisT[1]];
    const size_t volDimZ = volDim[axisT[2]];

    const int3 volDim_ = make_int3(volDim[0], volDim[1], volDim[2]);
    const int3 axisT_ = make_int3(axisT[0], axisT[1], axisT[2]);
    const int ySign = (invY ? -1 : 1);

    // setup block and grid
    const int blockSize = 8;
    const dim3 blockVolXZ(blockSize, blockSize, 1);
    const dim3 gridVolXZ(divUp(volDimX, blockVolXZ.x), divUp(volDimZ, blockVolXZ.y), 1);

    const int blockSizeL = 64;
    const dim3 blockColZ(blockSizeL, 1, 1);
    const dim3 gridColZ(divUp(volDimX, blockColZ.x), 1, 1);

    const dim3 blockVolSlide(blockSizeL, 1, 1);
    const dim3 gridVolSlide(divUp(volDimX, blockVolSlide.x), volDimZ, 1);

    CudaDeviceMemoryPitched<TSimAcc, 2> d_sliceBufferA(CudaSize<2>(volDimX, volDimZ));
    CudaDeviceMemoryPitched<TSimAcc, 2> d_sliceBufferB(CudaSize<2>(volDimX, volDimZ));

    CudaDeviceMemoryPitched<TSimAcc, 2>* d_xzSliceForY = &d_sliceBufferA; // Y slice
    CudaDeviceMemoryPitched<TSimAcc, 2>* d_xzSliceForYm1 = &d_sliceBufferB; // Y-1 slice

    CudaDeviceMemoryPitched<TSimAcc, 2> d_bestSimInYm1(CudaSize<2>(volDimX, 1)); // best sim score along the Y axis for each Z value

    // Copy the first XZ plane (at Y=0) from 'd_volSim' into 'd_xzSliceForYm1'
    volume_getVolumeXZSlice_kernel<TSimAcc, TSim><<<gridVolXZ, blockVolXZ>>>(
        d_xzSliceForYm1->getBuffer(),
        d_xzSliceForYm1->getPitch(),
        d_volSim.getBuffer(),
        d_volSim.getBytesPaddedUpToDim(1),
        d_volSim.getBytesPaddedUpToDim(0),
        volDim_, axisT_, 0); // Y=0

    // Set the first Z plane from 'd_volAgr' to 255
    volume_initVolumeYSlice_kernel<TSim><<<gridVolXZ, blockVolXZ>>>(
        d_volAgr.getBuffer(),
        d_volAgr.getBytesPaddedUpToDim(1),
        d_volAgr.getBytesPaddedUpToDim(0),
        volDim_, axisT_, 0, 255);

    for(int iy = 1; iy < volDimY; ++iy)
    {
        const int y = invY ? volDimY - 1 - iy : iy;

        // For each column: compute the best score
        // Foreach x:
        //   d_zBestSimInYm1[x] = min(d_xzSliceForY[1:height])
        volume_computeBestZInSlice_kernel<<<gridColZ, blockColZ>>>(
            d_xzSliceForYm1->getBuffer(), d_xzSliceForYm1->getPitch(),
            d_bestSimInYm1.getBuffer(),
            volDimX, volDimZ);

        // Copy the 'z' plane from 'd_volSimT' into 'd_xzSliceForY'
        volume_getVolumeXZSlice_kernel<TSimAcc, TSim><<<gridVolXZ, blockVolXZ>>>(
            d_xzSliceForY->getBuffer(),
            d_xzSliceForY->getPitch(),
            d_volSim.getBuffer(),
            d_volSim.getBytesPaddedUpToDim(1),
            d_volSim.getBytesPaddedUpToDim(0),
            volDim_, axisT_, y);

        volume_agregateCostVolumeAtXinSlices_kernel<<<gridVolSlide, blockVolSlide>>>(
            rc_tex,
            d_xzSliceForY->getBuffer(), d_xzSliceForY->getPitch(),              // inout: xzSliceForY
            d_xzSliceForYm1->getBuffer(), d_xzSliceForYm1->getPitch(),          // in:    xzSliceForYm1
            d_bestSimInYm1.getBuffer(),                                         // in:    bestSimInYm1
            d_volAgr.getBuffer(), d_volAgr.getBytesPaddedUpToDim(1), d_volAgr.getBytesPaddedUpToDim(0), // out:   volAgr
            volDim_, axisT_, 
            sgmParams.stepXY, 
            y, 
            sgmParams.p1, 
            sgmParams.p2Weighting,
            ySign, filteringIndex);

        std::swap(d_xzSliceForYm1, d_xzSliceForY);
    }
    // CHECK_CUDA_ERROR();
}

void ps_SGMretrieveBestDepth(int rcamCacheId,
    CudaDeviceMemoryPitched<float, 2>& bestDepth_dmp,
    CudaDeviceMemoryPitched<float, 2>& bestSim_dmp, 
    const CudaDeviceMemoryPitched<TSim, 3>& volSim_dmp,
    const CudaSize<3>& volDim,
    const CudaDeviceMemory<float>& depths_d,
    int scaleStep, bool interpolate)
{
  const int block_size = 8;
  const dim3 block(block_size, block_size, 1);
  const dim3 grid(divUp(volDim.x(), block_size), divUp(volDim.y(), block_size), 1);

  volume_retrieveBestZ_kernel<<<grid, block>>>(
    rcamCacheId,
    bestDepth_dmp.getBuffer(),
    bestDepth_dmp.getBytesPaddedUpToDim(0),
    bestSim_dmp.getBuffer(),
    bestSim_dmp.getBytesPaddedUpToDim(0),
    volSim_dmp.getBuffer(),
    volSim_dmp.getBytesPaddedUpToDim(1), volSim_dmp.getBytesPaddedUpToDim(0), 
    int(volDim.x()), 
    int(volDim.y()), 
    int(volDim.z()), 
    depths_d.getBuffer(),
    scaleStep,
    interpolate);
}



namespace ps
{
/*
 * static private variables in this class
 */
bool SimilarityVolume::_configured = false;
dim3 SimilarityVolume::_block( 32, 1, 1 ); // minimal default settings

SimilarityVolume::SimilarityVolume( const CudaSize<3>& volDim,
                                    int volStepXY,
                                    int scale,
                                    const std::vector<float>& depths_h)
    : _dimX(int(volDim.x()))
    , _dimY(int(volDim.y()))
    , _dimZ(int(volDim.z()))
    , _stepXY(volStepXY)
    , _scale(scale)
    , _depths_d(depths_h.data(), depths_h.size())
    , _stream_max( 2 )
{
    configureGrid();

    _sweep_stream.resize(_stream_max);
    for( hipStream_t& stream : _sweep_stream )
    {
        hipError_t err;
        err = hipStreamCreate( &stream );
        if( err != hipSuccess )
        {
            ALICEVISION_CU_PRINT_DEBUG("Failed to create a CUDA stream object for SimilarityVolume");
            stream = 0;
        }
    }
}

SimilarityVolume::~SimilarityVolume( )
{
    for( hipStream_t& stream : _sweep_stream )
    {
        hipStreamSynchronize( stream );
        if( stream != 0 ) hipStreamDestroy( stream );
    }
}

void SimilarityVolume::initOutputVolumes(
    CudaDeviceMemoryPitched<TSim, 3>& volBestSim_dmp,
    CudaDeviceMemoryPitched<TSim, 3>& volSecBestSim_dmp,
    const int streamIndex )
{
  const dim3 block(32, 4, 1);
  const dim3 grid(divUp(_dimX, block.x), divUp(_dimY, block.y), _dimZ);

  volume_init_kernel
    <<<grid, block, 0, SweepStream(streamIndex)>>>
    (volBestSim_dmp.getBuffer(),
      volBestSim_dmp.getBytesPaddedUpToDim(1),
      volBestSim_dmp.getBytesPaddedUpToDim(0),
      _dimX, _dimY);
  volume_init_kernel
    <<<grid, block, 0, SweepStream(streamIndex)>>>
    (volSecBestSim_dmp.getBuffer(),
      volSecBestSim_dmp.getBytesPaddedUpToDim(1),
      volSecBestSim_dmp.getBytesPaddedUpToDim(0),
      _dimX, _dimY);
}

void SimilarityVolume::compute(
                        CudaDeviceMemoryPitched<TSim, 3>& volBestSim_dmp,
                        CudaDeviceMemoryPitched<TSim, 3>& volSecBestSim_dmp,
                        const CameraStruct& rcam, int rcWidth, int rcHeight,
                        const CameraStruct& tcam, int tcWidth, int tcHeight,
                        const OneTC& cell,
                        const SgmParams& sgmParams,
                        const int streamIndex )
{
    TSim* gpu_volume_1st = volBestSim_dmp.getBuffer();
    TSim* gpu_volume_2nd = volSecBestSim_dmp.getBuffer();

    {
      const int startDepthIndex = cell.getDepthToStart();
      const int nbDepthsToSearch = cell.getDepthsToSearch();

      const dim3 grid(divUp(_dimX, _block.x), divUp(_dimY, _block.y), nbDepthsToSearch);

      ALICEVISION_CU_PRINT_DEBUG("====================");
      ALICEVISION_CU_PRINT_DEBUG("Volume slice kernel");
      ALICEVISION_CU_PRINT_DEBUG("RC: " << rcam.camId << ", TC: " << tcam.camId);
      ALICEVISION_CU_PRINT_DEBUG("Cell TC index: " << cell.getTCIndex());
      ALICEVISION_CU_PRINT_DEBUG("grid:  " << grid.x << ", " << grid.y << ", " << grid.z);
      ALICEVISION_CU_PRINT_DEBUG("block: " << _block.x << ", " << _block.y << ", " << _block.z);
      ALICEVISION_CU_PRINT_DEBUG("startDepthIndex: " << startDepthIndex);
      ALICEVISION_CU_PRINT_DEBUG("nbDepthsToSearch: " << nbDepthsToSearch);
      ALICEVISION_CU_PRINT_DEBUG("nb all depths: " << int(_depths_d.getUnitsTotal()));
      ALICEVISION_CU_PRINT_DEBUG("startDepthIndex+nbDepthsToSearch: " << startDepthIndex+nbDepthsToSearch);
      ALICEVISION_CU_PRINT_DEBUG("_dimX: " << _dimX << ", _dimY: " << _dimY);
      ALICEVISION_CU_PRINT_DEBUG("scale-1: " << prevScale() );
      ALICEVISION_CU_PRINT_DEBUG("rcWH / scale: " << rcWidth / _scale << "x" << rcHeight / _scale);
      ALICEVISION_CU_PRINT_DEBUG("tcWH / scale: " << tcWidth / _scale << "x" << tcHeight / _scale);
      ALICEVISION_CU_PRINT_DEBUG("====================");

      const Pyramid& rc_pyramid = *rcam.pyramid;
      const Pyramid& tc_pyramid = *tcam.pyramid;
      hipTextureObject_t rc_tex = rc_pyramid[prevScale()].tex;
      hipTextureObject_t tc_tex = tc_pyramid[prevScale()].tex;

      volume_slice_kernel
            <<<grid, _block, 0, SweepStream(streamIndex)>>>
            ( rc_tex,
              tc_tex,
              rcam.param_dev.i,
              tcam.param_dev.i,
              _depths_d.getBuffer(),
              startDepthIndex,
              nbDepthsToSearch,
              rcWidth / _scale, rcHeight / _scale,
              tcWidth / _scale, tcHeight / _scale,
              sgmParams.wsh,
              float(sgmParams.gammaC), 
              float(sgmParams.gammaP),
              gpu_volume_1st,
              volBestSim_dmp.getBytesPaddedUpToDim(1),
              volBestSim_dmp.getBytesPaddedUpToDim(0),
              gpu_volume_2nd,
              volSecBestSim_dmp.getBytesPaddedUpToDim(1),
              volSecBestSim_dmp.getBytesPaddedUpToDim(0),
              _stepXY,
              _dimX, _dimY);

        // hipDeviceSynchronize();
        // CHECK_CUDA_ERROR();
    }

    // hipDeviceSynchronize();
}

hipStream_t SimilarityVolume::SweepStream( int streamIndex )
{
    streamIndex %= _stream_max;
    return _sweep_stream[streamIndex];
}

void SimilarityVolume::WaitSweepStream( const int streamIndex )
{
    hipStreamSynchronize( SweepStream(streamIndex) );
}

void SimilarityVolume::configureGrid( )
{

    if( _configured ) return;
    _configured = true;

    int recommendedMinGridSize;
    int recommendedBlockSize;
    hipError_t err;
    err = hipOccupancyMaxPotentialBlockSize( &recommendedMinGridSize,
                                              &recommendedBlockSize,
                                              volume_slice_kernel,
                                              0, // dynamic shared mem size: none used
                                              0 ); // no block size limit, 1 thread OK
    if( err != hipSuccess )
    {
        ALICEVISION_CU_PRINT_DEBUG( "hipOccupancyMaxPotentialBlockSize failed for kernel volume_slice_kernel, using defaults" );
    }
    else
    {
        if( recommendedBlockSize > 32 )
        {
            _block.x = 32;
            _block.y = divUp( recommendedBlockSize, 32 );
        }
    }
}
}; // namespace ps

void ps_refineRcDepthMap(const CameraStruct& rcam, 
                         const CameraStruct& tcam,
                         float* inout_depthMap_hmh,
                         float* out_simMap_hmh,
                         int rcWidth, int rcHeight,
                         int tcWidth, int tcHeight,
                         const RefineParams& refineParams, 
                         int xFrom, int wPart, int CUDAdeviceNo)
{
    // setup block and grid
    const dim3 block(16, 16, 1);
    const dim3 grid(divUp(wPart, block.x), divUp(rcHeight, block.y), 1);

    const Pyramid& rcPyramid = *rcam.pyramid;
    const Pyramid& tcPyramid = *tcam.pyramid;
    const size_t pyramidScaleIndex = size_t(refineParams.scale) - 1;

    hipTextureObject_t rc_tex = rcPyramid[pyramidScaleIndex].tex;
    hipTextureObject_t tc_tex = tcPyramid[pyramidScaleIndex].tex;

    CudaDeviceMemoryPitched<float, 2> rcDepthMap_dmp(CudaSize<2>(wPart, rcHeight));
    copy(rcDepthMap_dmp, inout_depthMap_hmh, wPart, rcHeight);

    CudaDeviceMemoryPitched<float, 2> bestSimMap_dmp(CudaSize<2>(wPart, rcHeight));
    CudaDeviceMemoryPitched<float, 2> bestDptMap_dmp(CudaSize<2>(wPart, rcHeight));

    const int halfNSteps = ((refineParams.nDepthsToRefine - 1) / 2) + 1; // Default ntcsteps = 31

    for(int i = 0; i < halfNSteps; ++i)
    {
        refine_compUpdateYKNCCSimMapPatch_kernel<<<grid, block>>>(
            rcam.param_dev.i,
            tcam.param_dev.i,
            rc_tex, tc_tex,
            bestSimMap_dmp.getBuffer(), bestSimMap_dmp.getPitch(),
            bestDptMap_dmp.getBuffer(), bestDptMap_dmp.getPitch(),
            rcDepthMap_dmp.getBuffer(), rcDepthMap_dmp.getPitch(), 
            wPart, rcHeight, 
            refineParams.wsh, 
            refineParams.gammaC, 
            refineParams.gammaP,
            float(i), 
            refineParams.useTcOrRcPixSize, 
            xFrom,
            rcWidth, rcHeight,
            tcWidth, tcHeight);
    }

    for(int i = 1; i < halfNSteps; ++i)
    {
        refine_compUpdateYKNCCSimMapPatch_kernel<<<grid, block>>>(
            rcam.param_dev.i, 
            tcam.param_dev.i, 
            rc_tex, tc_tex, 
            bestSimMap_dmp.getBuffer(), bestSimMap_dmp.getPitch(), 
            bestDptMap_dmp.getBuffer(), bestDptMap_dmp.getPitch(),
            rcDepthMap_dmp.getBuffer(), rcDepthMap_dmp.getPitch(), 
            wPart, rcHeight, 
            refineParams.wsh,
            refineParams.gammaC, 
            refineParams.gammaP,
            float(-i),
            refineParams.useTcOrRcPixSize, 
            xFrom, 
            rcWidth, rcHeight, 
            tcWidth, tcHeight);
    }

    /*
    // Filter intermediate refined images does not improve
    for (int i = 0; i < 5; ++i)
    {
        // Filter refined depth map
        CudaTexture<float> depthTex(bestDptMap_dmp);
        float euclideanDelta = 1.0;
        int radius = 3;
        ps_bilateralFilter<float>(
            depthTex.textureObj,
            bestDptMap_dmp,
            euclideanDelta,
            radius);
        ps_medianFilter<float>(
            depthTex.textureObj,
            bestDptMap_dmp,
            radius);
    }
    */

    CudaDeviceMemoryPitched<float3, 2> lastThreeSimsMap_dmp(CudaSize<2>(wPart, rcHeight));
    CudaDeviceMemoryPitched<float, 2> simMap_dmp(CudaSize<2>(wPart, rcHeight));

    {
        // Set best sim map into lastThreeSimsMap_dmp.y
        refine_setLastThreeSimsMap_kernel<<<grid, block>>>(
            lastThreeSimsMap_dmp.getBuffer(), lastThreeSimsMap_dmp.getPitch(),
            bestSimMap_dmp.getBuffer(), bestSimMap_dmp.getPitch(), 
            wPart, rcHeight, 1);
        /*
        // Compute NCC for depth-1
        refine_compYKNCCSimMapPatch_kernel<<<grid, block>>>(
            rc_cam.param_dev.i, 
            tc_cam.param_dev.i,
            rc_tex, tc_tex,
            simMap_dmp.getBuffer(), simMap_dmp.getPitch(),
            bestDptMap_dmp.getBuffer(), bestDptMap_dmp.getPitch(),
            wPart, rcHeight,
            refineParams.wsh,
            refineParams.gammaC,
            refineParams.gammaP,
            0.0f, 
            refineParams.useTcOrRcPixSize, 
            xFrom,
            rcWidth, rcHeight,
            tcWidth, tcHeight);

        // Set sim for depth-1 into lastThreeSimsMap_dmp.y
        refine_setLastThreeSimsMap_kernel <<<grid, block>>>(
            lastThreeSimsMap_dmp.getBuffer(), lastThreeSimsMap_dmp.getPitch(),
            simMap_dmp.getBuffer(), simMap_dmp.getPitch(),
            wPart, rcHeight, 1);
        */
    }

    {
        // Compute NCC for depth-1
        refine_compYKNCCSimMapPatch_kernel<<<grid, block>>>(
            rcam.param_dev.i,
            tcam.param_dev.i, 
            rc_tex, tc_tex,
            simMap_dmp.getBuffer(), simMap_dmp.getPitch(),
            bestDptMap_dmp.getBuffer(), bestDptMap_dmp.getPitch(), 
            wPart, rcHeight, 
            refineParams.wsh,
            refineParams.gammaC, 
            refineParams.gammaP,
            -1.0f, 
            refineParams.useTcOrRcPixSize, 
            xFrom,
            rcWidth, rcHeight,
            tcWidth, tcHeight);

        // Set sim for depth-1 into lastThreeSimsMap_dmp.x
        refine_setLastThreeSimsMap_kernel<<<grid, block>>>(
            lastThreeSimsMap_dmp.getBuffer(), lastThreeSimsMap_dmp.getPitch(),
            simMap_dmp.getBuffer(), simMap_dmp.getPitch(), 
            wPart, rcHeight, 0);
    }

    {
        // Compute NCC for depth+1
        refine_compYKNCCSimMapPatch_kernel<<<grid, block>>>(
            rcam.param_dev.i,
            tcam.param_dev.i,
            rc_tex, tc_tex,
            simMap_dmp.getBuffer(), simMap_dmp.getPitch(),
            bestDptMap_dmp.getBuffer(), bestDptMap_dmp.getPitch(), 
            wPart, rcHeight, 
            refineParams.wsh,
            refineParams.gammaC, 
            refineParams.gammaP,
            +1.0f, 
            refineParams.useTcOrRcPixSize, 
            xFrom,
            rcWidth, rcHeight,
            tcWidth, tcHeight);

        // Set sim for depth+1 into lastThreeSimsMap_dmp.z
        refine_setLastThreeSimsMap_kernel<<<grid, block>>>(
            lastThreeSimsMap_dmp.getBuffer(), lastThreeSimsMap_dmp.getPitch(), 
            simMap_dmp.getBuffer(), simMap_dmp.getPitch(),
            wPart, rcHeight, 2);
    }

    // Interpolation from the lastThreeSimsMap_dmp
    refine_computeDepthSimMapFromLastThreeSimsMap_kernel<<<grid, block>>>(
        rcam.param_dev.i,
        tcam.param_dev.i,
        bestSimMap_dmp.getBuffer(), bestSimMap_dmp.getPitch(),
        bestDptMap_dmp.getBuffer(), bestDptMap_dmp.getPitch(),
        lastThreeSimsMap_dmp.getBuffer(), lastThreeSimsMap_dmp.getPitch(), 
        wPart, rcHeight,  
        refineParams.useTcOrRcPixSize, 
        xFrom);

    copy(out_simMap_hmh, wPart, rcHeight, bestSimMap_dmp);
    copy(inout_depthMap_hmh, wPart, rcHeight, bestDptMap_dmp);
}

/**
 * @brief ps_fuseDepthSimMapsGaussianKernelVoting
 * @param ndepthSimMaps: number of Tc cameras
 * @param nSamplesHalf (default value 150)
 * @param nDepthsToRefine (default value 31)
 */
void ps_fuseDepthSimMapsGaussianKernelVoting(int width, int height,
                                             CudaHostMemoryHeap<float2, 2>* out_depthSimMap_hmh,
                                             std::vector<CudaHostMemoryHeap<float2, 2>*>& depthSimMaps_hmh, 
                                             int ndepthSimMaps, 
                                             const RefineParams& refineParams)
{
    const float samplesPerPixSize = float(refineParams.nSamplesHalf / ((refineParams.nDepthsToRefine - 1) / 2));
    const float twoTimesSigmaPowerTwo = 2.0f * refineParams.sigma * refineParams.sigma;

    // setup block and grid
    const int block_size = 16;
    const dim3 block(block_size, block_size, 1);
    const dim3 grid(divUp(width, block_size), divUp(height, block_size), 1);

    CudaDeviceMemoryPitched<float2, 2> bestDepthSimMap_dmp(CudaSize<2>(width, height));
    CudaDeviceMemoryPitched<float2, 2> bestGsvSampleMap_dmp(CudaSize<2>(width, height));
    CudaDeviceMemoryPitched<float, 2> gsvSampleMap_dmp(CudaSize<2>(width, height));
    std::vector<CudaDeviceMemoryPitched<float2, 2>*> depthSimMaps_dmp(ndepthSimMaps);

    for(int i = 0; i < ndepthSimMaps; i++)
    {
        depthSimMaps_dmp[i] = new CudaDeviceMemoryPitched<float2, 2>(CudaSize<2>(width, height));
        copy((*depthSimMaps_dmp[i]), (*depthSimMaps_hmh[i]));
    }

    for(int s = -refineParams.nSamplesHalf; s <= refineParams.nSamplesHalf; s++) // (-150, 150)
    {
        for(int c = 1; c < ndepthSimMaps; c++) // number of T cameras
        {
            fuse_computeGaussianKernelVotingSampleMap_kernel<<<grid, block>>>(
                gsvSampleMap_dmp.getBuffer(), gsvSampleMap_dmp.getPitch(),
                depthSimMaps_dmp[c]->getBuffer(), depthSimMaps_dmp[c]->getPitch(),
                depthSimMaps_dmp[0]->getBuffer(), depthSimMaps_dmp[0]->getPitch(),
                width, height, (float)s, c - 1, samplesPerPixSize, twoTimesSigmaPowerTwo);
        }
        fuse_updateBestGaussianKernelVotingSampleMap_kernel<<<grid, block>>>(
            bestGsvSampleMap_dmp.getBuffer(), bestGsvSampleMap_dmp.getPitch(),
            gsvSampleMap_dmp.getBuffer(), gsvSampleMap_dmp.getPitch(), 
            width, height, (float)s, s + refineParams.nSamplesHalf);
    }

    fuse_computeFusedDepthSimMapFromBestGaussianKernelVotingSampleMap_kernel<<<grid, block>>>(
        bestDepthSimMap_dmp.getBuffer(), bestDepthSimMap_dmp.getPitch(),
        bestGsvSampleMap_dmp.getBuffer(), bestGsvSampleMap_dmp.getPitch(),
        depthSimMaps_dmp[0]->getBuffer(), depthSimMaps_dmp[0]->getPitch(), 
        width, height, samplesPerPixSize);

    copy((*out_depthSimMap_hmh), bestDepthSimMap_dmp);

    for(int i = 0; i < ndepthSimMaps; i++)
    {
        delete depthSimMaps_dmp[i];
    }
}

void ps_optimizeDepthSimMapGradientDescent(const CameraStruct& rcam,
                                           CudaHostMemoryHeap<float2, 2>& out_optimizedDepthSimMap_hmh,
                                           const CudaHostMemoryHeap<float2, 2>& sgmDepthPixSizeMap_hmh,
                                           const CudaHostMemoryHeap<float2, 2>& refinedDepthSimMap_hmh,
                                           const CudaSize<2>& depthSimMapPartDim, 
                                           const RefineParams& refineParams,
                                           int CUDAdeviceNo, int nbCamsAllocated, int yFrom)
{
    const int partWidth = depthSimMapPartDim.x();
    const int partHeight = depthSimMapPartDim.y(); 
    const float samplesPerPixSize = float(refineParams.nSamplesHalf / ((refineParams.nDepthsToRefine - 1) / 2));

    // setup block and grid
    const int block_size = 16;
    const dim3 block(block_size, block_size, 1);
    const dim3 grid(divUp(partWidth, block_size), divUp(partHeight, block_size), 1);

    const CudaDeviceMemoryPitched<float2, 2> sgmDepthPixSizeMap_dmp(sgmDepthPixSizeMap_hmh);
    const CudaDeviceMemoryPitched<float2, 2> refinedDepthSimMap_dmp(refinedDepthSimMap_hmh);

    CudaDeviceMemoryPitched<float, 2> optDepthMap_dmp(depthSimMapPartDim);
    CudaDeviceMemoryPitched<float2, 2> optDepthSimMap_dmp(depthSimMapPartDim);
    copy(optDepthSimMap_dmp, sgmDepthPixSizeMap_dmp);

    // get rc CUDA texture object
    const size_t pyramidScaleIndex = size_t(refineParams.scale) - 1;
    const Pyramid& rcPyramid = *rcam.pyramid;
    hipTextureObject_t rc_tex = rcPyramid[pyramidScaleIndex].tex;

    CudaDeviceMemoryPitched<float, 2> imgVariance_dmp(depthSimMapPartDim);
    {
        const dim3 lblock(32, 2, 1);
        const dim3 lgrid(divUp(partWidth, lblock.x), divUp(partHeight, lblock.y), 1);

        compute_varLofLABtoW_kernel<<<lgrid, lblock>>>(rc_tex,
                                                       imgVariance_dmp.getBuffer(), 
                                                       imgVariance_dmp.getPitch(),
                                                       partWidth, partHeight, yFrom);
    }
    CudaTexture<float> imgVarianceTex(imgVariance_dmp);

    for(int iter = 0; iter < refineParams.nIters; ++iter) // nIters: 100 by default
    {
        // Copy depths values from optDepthSimMap to optDepthMap
        fuse_getOptDeptMapFromOptDepthSimMap_kernel<<<grid, block>>>(optDepthMap_dmp.getBuffer(), optDepthMap_dmp.getPitch(),
                                                                     optDepthSimMap_dmp.getBuffer(), optDepthSimMap_dmp.getPitch(), 
                                                                     partWidth, partHeight);

        CudaTexture<float> depthTex(optDepthMap_dmp);

        // Adjust depth/sim by using previously computed depths
        fuse_optimizeDepthSimMap_kernel<<<grid, block>>>(rc_tex, 
                                                         rcam.param_dev.i,
                                                         imgVarianceTex.textureObj, 
                                                         depthTex.textureObj,
                                                         optDepthSimMap_dmp.getBuffer(), optDepthSimMap_dmp.getPitch(),
                                                         sgmDepthPixSizeMap_dmp.getBuffer(), sgmDepthPixSizeMap_dmp.getPitch(),
                                                         refinedDepthSimMap_dmp.getBuffer(), refinedDepthSimMap_dmp.getPitch(), 
                                                         partWidth, partHeight, iter, samplesPerPixSize, yFrom);
    }

    copy(out_optimizedDepthSimMap_hmh, optDepthSimMap_dmp);
}

// uchar4 with 0..255 components => float3 with 0..1 components
inline __device__ __host__ float3 uchar4_to_float3(const uchar4 c)
{
    return make_float3(float(c.x) / 255.0f, float(c.y) / 255.0f, float(c.z) / 255.0f);
}

void ps_getSilhoueteMap(CudaHostMemoryHeap<bool, 2>* omap_hmh, int width,
                        int height, int scale,
                        int step,
                        CameraStruct& cam,
                        uchar4 maskColorRgb, bool verbose)
{
    clock_t tall = tic();

    uchar4 maskColorLab;
    float3 flab = xyz2lab(h_rgb2xyz(uchar4_to_float3(maskColorRgb)));
    maskColorLab.x = (unsigned char)(flab.x);
    maskColorLab.y = (unsigned char)(flab.y);
    maskColorLab.z = (unsigned char)(flab.z);
    maskColorLab.w = 0;

    // setup block and grid
    int block_size = 16;
    dim3 block(block_size, block_size, 1);
    dim3 grid(divUp(width / step, block_size), divUp(height / step, block_size), 1);

    Pyramid& pyramid = *cam.pyramid;

    CudaDeviceMemoryPitched<bool, 2> map_dmp(CudaSize<2>(width / step, height / step));
    getSilhoueteMap_kernel<<<grid, block>>>(
        pyramid[scale].tex,
        map_dmp.getBuffer(), map_dmp.getPitch(),
        step, width, height, maskColorLab);
    CHECK_CUDA_ERROR();

    copy((*omap_hmh), map_dmp);

    if(verbose)
        printf("gpu elapsed time: %f ms \n", toc(tall));
}


void ps_loadCameraStructs( const CameraStructBase* hst,
                           const CamCacheIdx&      offset,
                           hipStream_t            stream )
{
    hipMemcpyKind kind = hipMemcpyHostToDevice;
    hipError_t err;
    if( stream == 0 )
    {
        err = hipMemcpyToSymbol(HIP_SYMBOL( camsBasesDev),
                                  &hst[offset.i],
                                  sizeof(CameraStructBase),
                                  offset.i*sizeof(CameraStructBase),
                                  kind );
    }
    else
    {
        err = hipMemcpyToSymbolAsync(HIP_SYMBOL( camsBasesDev),
                                       &hst[offset.i],
                                       sizeof(CameraStructBase),
                                       offset.i*sizeof(CameraStructBase),
                                       kind,
                                       stream );
    }
    THROW_ON_CUDA_ERROR( err, "Failed to copy CameraStructs from host to device in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) );
}

} // namespace depthMap
} // namespace aliceVision
