#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2017 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#ifndef ALICEVISION_CUDA_deviceCommon_device_global_cu
#define ALICEVISION_CUDA_deviceCommon_device_global_cu

#include <aliceVision/depthMap/cuda/commonStructures.hpp>

namespace aliceVision {
namespace depthMap {

// Helper functions
// function clamping x between a and b
__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}


////////////////////////////////////////////////////////////////////////////////
// CONSTANT MEMORY

// MATLAB: x = [-2:2]; delta = 1; y = exp( - (x .* x) / (2 * delta * delta)); format long g; y
__constant__ float gauss5[5] = {0.135335283236613f, 0.606530659712633f, 1.0f, 0.606530659712633f,
                                           0.135335283236613f};
__constant__ float sumGauss55 = 6.16892408102888f;

// MATLAB: distFcnHeight=1.0; maxDist = 0.3;  dist = 0:0.01:1; y =
// 1-distFcnHeight*exp(-(dist.*dist)/(2*maxDist*maxDist)); plot(dist,y);
// MATLAB: distFcnHeight=1.0; maxDist = 0.3;  dist = 0:0.25:1; y =
// 1-distFcnHeight*exp(-(dist.*dist)/(2*maxDist*maxDist)); plot(dist,y); int32(125*y)
__constant__ unsigned char distFcnConst5[5] = {0, 37, 94, 120, 125};

// MATLAB: distFcnHeight=1.0; maxDist = 0.3;  dist = 0:1/2:1; y =
// 1-distFcnHeight*exp(-(dist.*dist)/(2*maxDist*maxDist)); plot(dist,y); int32(125*y)
__constant__ unsigned char distFcnConst3[3] = {0, 94, 125};

__constant__ CameraStructBase camsBasesDev[MAX_CONSTANT_CAMERA_PARAM_SETS];


} // namespace depthMap
} // namespace aliceVision

#else // ALICEVISION_CUDA_deviceCommon_device_global_cu
#error "deviceCommon/device_global.cu has been included twice"
#endif // ALICEVISION_CUDA_deviceCommon_device_global_cu
