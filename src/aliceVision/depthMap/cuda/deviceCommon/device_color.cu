#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2017 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#pragma once

#include <aliceVision/depthMap/cuda/deviceCommon/device_utils.cuh>

namespace aliceVision {
namespace depthMap {

inline __device__ float Euclidean(const float3 x1, const float3 x2)
{
    // return sqrtf((x1.x - x2.x) * (x1.x - x2.x) + (x1.y - x2.y) * (x1.y - x2.y) + (x1.z - x2.z) * (x1.z - x2.z));
    return norm3df(x1.x - x2.x, x1.y - x2.y, x1.z - x2.z);
}

inline __device__ float Euclidean3(const float4 x1, const float4 x2)
{
    // return sqrtf((x1.x - x2.x) * (x1.x - x2.x) + (x1.y - x2.y) * (x1.y - x2.y) + (x1.z - x2.z) * (x1.z - x2.z));
    return norm3df(x1.x - x2.x, x1.y - x2.y, x1.z - x2.z);
}

//== colour conversion utils ======================================================================

// sRGB (0..1) to linear RGB (0..1)
inline __device__ float3 srgb2rgb(const float3 c)
{
    return make_float3(c.x <= 0.04045f ? c.x / 12.92f : __powf((c.x + 0.055f) / 1.055f, 2.4f),
                       c.y <= 0.04045f ? c.y / 12.92f : __powf((c.y + 0.055f) / 1.055f, 2.4f),
                       c.z <= 0.04045f ? c.z / 12.92f : __powf((c.z + 0.055f) / 1.055f, 2.4f));
}

// linear RGB (0..1) to XZY (0..1) using sRGB primaries
inline __device__ float3 rgb2xyz(const float3 c)
{
    return make_float3(0.4124564f * c.x + 0.3575761f * c.y + 0.1804375f * c.z,
                       0.2126729f * c.x + 0.7151522f * c.y + 0.0721750f * c.z,
                       0.0193339f * c.x + 0.1191920f * c.y + 0.9503041f * c.z);
}

inline __host__ float3 h_rgb2xyz(const float3 c)
{
    return make_float3(0.4124564f * c.x + 0.3575761f * c.y + 0.1804375f * c.z,
                       0.2126729f * c.x + 0.7151522f * c.y + 0.0721750f * c.z,
                       0.0193339f * c.x + 0.1191920f * c.y + 0.9503041f * c.z);
}

// linear RGB (0..1) to HSL (0..1)
inline __device__ float3 rgb2hsl(const float3& c)
{
    const float cmin = fminf(c.x, fminf(c.y, c.z));
    const float cmax = fmaxf(c.x, fmaxf(c.y, c.z));

    float h = 0.0f;
    if(cmin == cmax)
    { /* h = 0.0f; */
    }
    else if(cmax == c.x)
    {
        h = ((c.y - c.z) / (cmax - cmin) + 6.0f) / 6.0f;
        if(h >= 1.0f)
            h -= 1.0f;
    }
    else if(cmax == c.y)
    {
        h = ((c.z - c.x) / (cmax - cmin) + 2.0f) / 6.0f;
    }
    else /* if(cmax == c.z) */
    {
        h = ((c.x - c.y) / (cmax - cmin) + 4.0f) / 6.0f;
    }

    const float l = 0.5f * (cmin + cmax);

    float s = 0.0f;
    if(cmin == cmax)
    { /* s = 0.0f; */
    }
    else if(l <= 0.5f)
    {
        s = (cmax - cmin) / (2.0f * l);
    }
    else /* if(l > 0.5f) */
    {
        s = (cmax - cmin) / (2.0f - 2.0f * l);
    }

    return make_float3(h, s, l);
}

// XYZ (0..1) to CIELAB (0..255) assuming D65 whitepoint
inline __host__ __device__ float3 xyz2lab(const float3 c)
{
    // assuming whitepoint D65, XYZ=(0.95047, 1.00000, 1.08883)
    float3 r = make_float3(c.x / 0.95047f, c.y, c.z / 1.08883f);

    float3 f = make_float3((r.x > 216.0f / 24389.0f ? cbrtf(r.x) : (24389.0f / 27.0f * r.x + 16.0f) / 116.0f),
                           (r.y > 216.0f / 24389.0f ? cbrtf(r.y) : (24389.0f / 27.0f * r.y + 16.0f) / 116.0f),
                           (r.z > 216.0f / 24389.0f ? cbrtf(r.z) : (24389.0f / 27.0f * r.z + 16.0f) / 116.0f));

    float3 out = make_float3(116.0f * f.y - 16.0f, 500.0f * (f.x - f.y), 200.0f * (f.y - f.z));

    // convert values to fit into 0..255 (could be out-of-range)
    // TODO FACA: use float textures, the values are out-of-range for a and b.
    out.x = out.x * 2.55f;
    out.y = out.y * 2.55f;
    out.z = out.z * 2.55f;
    return out;
}

inline __device__ float rgb2gray(const uchar4 c)
{
    return 0.2989f * (float)c.x + 0.5870f * (float)c.y + 0.1140f * (float)c.z;
}

/**
 * @brief 
 * 
 * "Adaptive Support-Weight Approach for Correspondence Search", Kuk-Jin Yoon, In So Kweon
 * http://koasas.kaist.ac.kr/bitstream/10203/21048/1/000235253300014.pdf
 * 
 * @param[in] dx
 * @param[in] dy
 * @param[in] c1
 * @param[in] c2
 * @param[in] gammaC Strength of Grouping by Color similarity 5.5 / 105.5
 * @param[in] gammaP Strength of Grouping by Proximity          8 / 4
 * @return distance value
 */
inline __device__ float CostYKfromLab(const int dx, const int dy, const float4 c1, const float4 c2, const float gammaC,
                                      const float gammaP)
{
    // const float deltaC = 0; // ignore colour difference

    //// AD in RGB
    // const float deltaC =
    //    fabsf(float(c1.x) - float(c2.x)) +
    //    fabsf(float(c1.y) - float(c2.y)) +
    //    fabsf(float(c1.z) - float(c2.z));

    //// Euclidean distance in RGB
    // const float deltaC = Euclidean(
    //    uchar4_to_float3(c1),
    //    uchar4_to_float3(c2)
    //);

    //// Euclidean distance in Lab, assuming sRGB
    // const float deltaC = Euclidean(
    //    xyz2lab(rgb2xyz(srgb2rgb(uchar4_to_float3(c1)))),
    //    xyz2lab(rgb2xyz(srgb2rgb(uchar4_to_float3(c2))))
    //);

    // Euclidean distance in Lab, assuming linear RGB
    float deltaC = Euclidean3(c1, c2);
    // const float deltaC = fmaxf(fabs(c1.x-c2.x),fmaxf(fabs(c1.y-c2.y),fabs(c1.z-c2.z)));

    deltaC /= gammaC;

    // spatial distance to the center of the patch (in pixels)
    float deltaP = sqrtf(float(dx * dx + dy * dy));

    deltaP /= gammaP;

    deltaC += deltaP;

    return __expf(-deltaC); // Yoon & Kweon
    // return __expf(-(deltaC * deltaC / (2 * gammaC * gammaC))) * sqrtf(__expf(-(deltaP * deltaP / (2 * gammaP * gammaP)))); // DCB
}
/*
inline __device__ float CostYKfromLab(const float4 c1, const float4 c2, const float gammaC)
{
    // Euclidean distance in Lab, assuming linear RGB
    const float deltaC = Euclidean3(c1, c2);
    // const float deltaC = fmaxf(fabs(c1.x-c2.x),fmaxf(fabs(c1.y-c2.y),fabs(c1.z-c2.z)));

    return __expf(-(deltaC / gammaC)); // Yoon & Kweon
}
*/
__global__ void rgb2lab_kernel(CudaRGBA* irgbaOlab, int irgbaOlab_p, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if((x >= width) || (y >= height))
        return;

    CudaRGBA* rgb = get2DBufferAt(irgbaOlab, irgbaOlab_p, x, y);
    float3 flab = xyz2lab(rgb2xyz(make_float3(rgb->x / 255.f, rgb->y / 255.f, rgb->z / 255.f)));

    rgb->x = flab.x;
    rgb->y = flab.y;
    rgb->z = flab.z;
}

/*
    Because a 2D gaussian mask is symmetry in row and column,
    here only generate a 1D mask, and use the product by row
    and column index later.

    1D gaussian distribution :
        g(x, d) -- C * exp(-x^2/d^2), C is a constant amplifier

    parameters:
    og - output gaussian array in global memory
    delta - the 2nd parameter 'd' in the above function
    radius - half of the filter size
             (total filter size = 2 * radius + 1)
*/
// use only one block

/*
__global__ void downscale_kernel(unsigned char* tex, int tex_p, int width, int height, int scale)
{
        int x = blockIdx.x*blockDim.x + threadIdx.x;
        int y = blockIdx.y*blockDim.y + threadIdx.y;

        if ((x<width)&&(y<height)) {
                tex[y*tex_p+x] = 255.0f*tex2D(rtex, (float)x*(float)scale+0.5f, (float)y*(float)scale+0.5f);
        };
}
*/

} // namespace depthMap
} // namespace aliceVision
