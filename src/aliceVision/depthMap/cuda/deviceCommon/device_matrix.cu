#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2017 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#pragma once

// mn MATRIX ADDRESSING: mxy = x*n+y (x-row,y-col), (m-number of rows, n-number of columns)

#include <hip/hip_math_constants.h>
#include <aliceVision/depthMap/cuda/deviceCommon/device_matrix.cuh>

namespace aliceVision {
namespace depthMap {

__device__ float2 project3DPoint( const float* M3x4, const float3& V)
{
    float3 p = M3x4mulV3(M3x4, V);
    return make_float2(p.x / p.z, p.y / p.z);
}

__device__ void M3x3mulM3x3(float* O3x3, const float* A3x3, const float* B3x3)
{
    O3x3[0] = A3x3[0] * B3x3[0] + A3x3[3] * B3x3[1] + A3x3[6] * B3x3[2];
    O3x3[3] = A3x3[0] * B3x3[3] + A3x3[3] * B3x3[4] + A3x3[6] * B3x3[5];
    O3x3[6] = A3x3[0] * B3x3[6] + A3x3[3] * B3x3[7] + A3x3[6] * B3x3[8];

    O3x3[1] = A3x3[1] * B3x3[0] + A3x3[4] * B3x3[1] + A3x3[7] * B3x3[2];
    O3x3[4] = A3x3[1] * B3x3[3] + A3x3[4] * B3x3[4] + A3x3[7] * B3x3[5];
    O3x3[7] = A3x3[1] * B3x3[6] + A3x3[4] * B3x3[7] + A3x3[7] * B3x3[8];

    O3x3[2] = A3x3[2] * B3x3[0] + A3x3[5] * B3x3[1] + A3x3[8] * B3x3[2];
    O3x3[5] = A3x3[2] * B3x3[3] + A3x3[5] * B3x3[4] + A3x3[8] * B3x3[5];
    O3x3[8] = A3x3[2] * B3x3[6] + A3x3[5] * B3x3[7] + A3x3[8] * B3x3[8];
}

__device__ void M3x3minusM3x3(float* O3x3, float* A3x3, float* B3x3)
{
    for(int i = 0; i < 9; i++)
    {
        O3x3[i] = A3x3[i] - B3x3[i];
    };
}

__device__ void M3x3transpose(float* O3x3, const float* A3x3)
{
    O3x3[0] = A3x3[0];
    O3x3[1] = A3x3[3];
    O3x3[2] = A3x3[6];
    O3x3[3] = A3x3[1];
    O3x3[4] = A3x3[4];
    O3x3[5] = A3x3[7];
    O3x3[6] = A3x3[2];
    O3x3[7] = A3x3[5];
    O3x3[8] = A3x3[8];
}

__device__ void outerMultiply(float* O3x3, const float3& a, const float3& b)
{
    O3x3[0] = a.x * b.x;
    O3x3[3] = a.x * b.y;
    O3x3[6] = a.x * b.z;
    O3x3[1] = a.y * b.x;
    O3x3[4] = a.y * b.y;
    O3x3[7] = a.y * b.z;
    O3x3[2] = a.z * b.x;
    O3x3[5] = a.z * b.y;
    O3x3[8] = a.z * b.z;
}

__device__ float3 linePlaneIntersect(const float3& linePoint, const float3& lineVect, const float3& planePoint,
                                     const float3& planeNormal)
{
    float k = (dot(planePoint, planeNormal) - dot(planeNormal, linePoint)) / dot(planeNormal, lineVect);
    return linePoint + lineVect * k;
}

__device__ float3 closestPointOnPlaneToPoint(const float3& point, const float3& planePoint,
                                             const float3& planeNormalNormalized)
{
    return point - planeNormalNormalized * dot(planeNormalNormalized, point - planePoint);
}

__device__ float3 closestPointToLine3D(const float3& point, const float3& linePoint, const float3& lineVectNormalized)
{
    return linePoint + lineVectNormalized * dot(lineVectNormalized, point - linePoint);
}

__device__ float pointLineDistance3D(const float3& point, const float3& linePoint, const float3& lineVectNormalized)
{
    return size(cross(lineVectNormalized, linePoint - point));
}

// v1,v2 dot not have to be normalized
__device__ float angleBetwV1andV2(const float3& iV1, const float3& iV2)
{
    float3 V1, V2;
    V1 = iV1;
    normalize(V1);
    V2 = iV2;
    normalize(V2);

    return fabsf(acosf(V1.x * V2.x + V1.y * V2.y + V1.z * V2.z) / (HIP_PI_F / 180.0f));
}

__device__ float angleBetwABandAC(const float3& A, const float3& B, const float3& C)
{
    float3 V1, V2;
    V1 = B - A;
    V2 = C - A;
    normalize(V1);
    normalize(V2);

    float a = acosf(V1.x * V2.x + V1.y * V2.y + V1.z * V2.z);
    a = isinf(a) ? 0.0f : a;

    return fabsf(a) / (HIP_PI_F / 180.0f);
}

__device__ float3 lineLineIntersect(float* k, float* l, float3* lli1, float3* lli2,
    const float3& p1, const float3& p2, const float3& p3, const float3& p4)
{
    /*
    %  [pa, pb, mua, mub] = LineLineIntersect(p1,p2,p3,p4)
    %
    %   Calculates the line segment pa_pb that is the shortest route
    %   between two lines p1_p2 and p3_p4. Calculates also the values of
    %   mua and mub where
    %        pa = p1 + mua (p2 - p1)
    %        pb = p3 + mub (p4 - p3)
    %
    %   Returns a MATLAB error if no solution exists.
    %
    %   This a simple conversion to MATLAB of the C code posted by Paul
    %   Bourke at
    %   http://astronomy.swin.edu.au/~pbourke/geometry/lineline3d/. The
    %   author of this all too imperfect translation is Cristian Dima
    %   (csd@cmu.edu)
    */

    float d1343, d4321, d1321, d4343, d2121, denom, numer, p13[3], p43[3], p21[3], pa[3], pb[3], muab[2];

    p13[0] = p1.x - p3.x;
    p13[1] = p1.y - p3.y;
    p13[2] = p1.z - p3.z;

    p43[0] = p4.x - p3.x;
    p43[1] = p4.y - p3.y;
    p43[2] = p4.z - p3.z;

    /*
    if ((abs(p43[0])  < eps) & ...
        (abs(p43[1])  < eps) & ...
        (abs(p43[2])  < eps))
      error('Could not compute LineLineIntersect!');
    end
    */

    p21[0] = p2.x - p1.x;
    p21[1] = p2.y - p1.y;
    p21[2] = p2.z - p1.z;

    /*
    if ((abs(p21[0])  < eps) & ...
        (abs(p21[1])  < eps) & ...
        (abs(p21[2])  < eps))
      error('Could not compute LineLineIntersect!');
    end
    */

    d1343 = p13[0] * p43[0] + p13[1] * p43[1] + p13[2] * p43[2];
    d4321 = p43[0] * p21[0] + p43[1] * p21[1] + p43[2] * p21[2];
    d1321 = p13[0] * p21[0] + p13[1] * p21[1] + p13[2] * p21[2];
    d4343 = p43[0] * p43[0] + p43[1] * p43[1] + p43[2] * p43[2];
    d2121 = p21[0] * p21[0] + p21[1] * p21[1] + p21[2] * p21[2];

    denom = d2121 * d4343 - d4321 * d4321;

    /*
    if (abs(denom) < eps)
      error('Could not compute LineLineIntersect!');
    end
     */

    numer = d1343 * d4321 - d1321 * d4343;

    muab[0] = numer / denom;
    muab[1] = (d1343 + d4321 * muab[0]) / d4343;

    pa[0] = p1.x + muab[0] * p21[0];
    pa[1] = p1.y + muab[0] * p21[1];
    pa[2] = p1.z + muab[0] * p21[2];

    pb[0] = p3.x + muab[1] * p43[0];
    pb[1] = p3.y + muab[1] * p43[1];
    pb[2] = p3.z + muab[1] * p43[2];

    float3 S;
    S.x = (pa[0] + pb[0]) / 2.0;
    S.y = (pa[1] + pb[1]) / 2.0;
    S.z = (pa[2] + pb[2]) / 2.0;

    *k = muab[0];
    *l = muab[1];

    lli1->x = pa[0];
    lli1->y = pa[1];
    lli1->z = pa[2];

    lli2->x = pb[0];
    lli2->y = pb[1];
    lli2->z = pb[2];

    return S;
}

/**
 * f(x) = min + (max-min) * \frac{1}{1 + e^{10 * (x - mid) / width}}
 * https://www.desmos.com/calculator/1qvampwbyx
 */
__device__ float sigmoid(float zeroVal, float endVal, float sigwidth, float sigMid, float xval)
{
    return zeroVal + (endVal - zeroVal) * (1.0f / (1.0f + expf(10.0f * ((xval - sigMid) / sigwidth))));
}

/**
 * f(x) = min + (max-min) * \frac{1}{1 + e^{10 * (mid - x) / width}}
 */
__device__ float sigmoid2(float zeroVal, float endVal, float sigwidth, float sigMid, float xval)
{
    return zeroVal + (endVal - zeroVal) * (1.0f / (1.0f + expf(10.0f * ((sigMid - xval) / sigwidth))));
}

} // namespace depthMap
} // namespace aliceVision
